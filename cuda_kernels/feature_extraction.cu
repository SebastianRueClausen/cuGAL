#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <utility>
#include "common.cuh"

template <typename index_t>
__device__ std::pair<int, int> edge_range(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    uint32_t vertex_index
) {
    return std::make_pair(
        row_pointers[vertex_index],
        row_pointers.size(0) == vertex_index + 1 ? col_indices.size(0) : row_pointers[vertex_index + 1]
    );
}

template <typename index_t>
__device__ int intersection_size(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    uint32_t vertex_a,
    uint32_t vertex_b
) {
    const auto [a_start, a_end] = edge_range(col_indices, row_pointers, vertex_a);
    const auto [b_start, b_end] = edge_range(col_indices, row_pointers, vertex_b);

    int intersection_size = 0;

    for (auto a_index = a_start, b_index = b_start; a_index < a_end && b_index < b_end;) {
        const auto a_edge = col_indices[a_index];
        const auto b_edge = col_indices[b_index];

        if (a_edge == b_edge) {
            intersection_size++;
            a_index++;
            b_index++;
        } else if (a_edge > b_edge) {
            b_index++;
        } else {
            a_index++;
        }
    }

    return intersection_size;
}

template <typename index_t>
__global__ void vertex_clustering(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    Accessor<float, 1> coeffs
) {
    const auto vertex_index = threadIdx.x + blockIdx.x * blockDim.x;

    if (vertex_index >= row_pointers.size(0)) {
        return;
    }

    auto triangle_count = 0;
    const auto [start, end] = edge_range(col_indices, row_pointers, vertex_index);
    for (auto edge_index = start; edge_index < end; edge_index++) {
        triangle_count += intersection_size(col_indices, row_pointers, vertex_index, col_indices[edge_index]);
    }

    const auto degree = end - start;
    coeffs[vertex_index] = triangle_count == 0 ? 0.0 : float(triangle_count) / (degree * (degree - 1));
}

void graph_clustering(torch::Tensor col_indices, torch::Tensor row_pointers, torch::Tensor coeffs) {
    constexpr auto block_size = 64;

    const auto thread_count = row_pointers.size(0);
    const auto block_count = div_ceil(thread_count, block_size);

    if (col_indices.scalar_type() == torch::ScalarType::Short) {
        vertex_clustering<short><<<block_count, block_size>>>(
            col_indices.packed_accessor32<short, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            coeffs.packed_accessor32<float,  1>()
        );
    } else if (col_indices.scalar_type() == torch::ScalarType::Int) {
        vertex_clustering<int><<<block_count, block_size>>>(
            col_indices.packed_accessor32<int, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            coeffs.packed_accessor32<float,  1>()
        );
    }

    hipDeviceSynchronize();
}
