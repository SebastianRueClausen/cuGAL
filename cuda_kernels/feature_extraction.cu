#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <utility>
#include "common.cuh"

__device__ std::pair<int, int> edge_range(
    uint32_t edge_count,
    const Accessor<int, 1> row_pointers,
    uint32_t vertex_index
) {
    return std::make_pair(
        row_pointers[vertex_index],
        row_pointers.size(0) == vertex_index + 1 ? edge_count : row_pointers[vertex_index + 1]
    );
}

template <typename index_t>
__device__ int intersection_size(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    uint32_t vertex_a,
    uint32_t vertex_b
) {
    const auto [a_start, a_end] = edge_range(col_indices.size(0), row_pointers, vertex_a);
    const auto [b_start, b_end] = edge_range(col_indices.size(0), row_pointers, vertex_b);

    int intersection_size = 0;

    for (auto a_index = a_start, b_index = b_start; a_index < a_end && b_index < b_end;) {
        const auto a_edge = col_indices[a_index];
        const auto b_edge = col_indices[b_index];

        if (a_edge == b_edge) {
            intersection_size++;
            a_index++;
            b_index++;
        } else if (a_edge > b_edge) {
            b_index++;
        } else {
            a_index++;
        }
    }

    return intersection_size;
}

template <typename index_t>
__global__ void vertex_features(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    Accessor<float, 1> clustering,
    Accessor<float, 1> degrees
) {
    const auto vertex_index = threadIdx.x + blockIdx.x * blockDim.x;

    if (vertex_index >= row_pointers.size(0)) {
        return;
    }

    auto triangle_count = 0;
    const auto [start, end] = edge_range(col_indices.size(0), row_pointers, vertex_index);
    for (auto edge_index = start; edge_index < end; edge_index++) {
        triangle_count += intersection_size(col_indices, row_pointers, vertex_index, col_indices[edge_index]);
    }

    const auto degree = end - start;
    degrees[vertex_index]  = float(degree);
    clustering[vertex_index] = triangle_count == 0 ? 0.0 : float(triangle_count) / (degree * (degree - 1));
}

void graph_features(torch::Tensor col_indices, torch::Tensor row_pointers, torch::Tensor clustering, torch::Tensor degrees) {
    constexpr auto block_size = 64;

    const auto thread_count = row_pointers.size(0);
    const auto block_count = div_ceil(thread_count, block_size);

    if (col_indices.scalar_type() == torch::ScalarType::Short) {
        vertex_features<short><<<block_count, block_size>>>(
            col_indices.packed_accessor32<short, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            clustering.packed_accessor32<float,  1>(),
            degrees.packed_accessor32<float,  1>()
        );
    } else if (col_indices.scalar_type() == torch::ScalarType::Int) {
        vertex_features<int><<<block_count, block_size>>>(
            col_indices.packed_accessor32<int, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            clustering.packed_accessor32<float,  1>(),
            degrees.packed_accessor32<float,  1>()
        );
    }

    hipDeviceSynchronize();
}

template <typename index_t>
__global__ void vertex_average_neighbor_features(
    const Accessor<index_t, 1> col_indices,
    const Accessor<int, 1> row_pointers,
    const Accessor<float, 1> features,
    Accessor<float, 1> averages
) {
    const auto vertex_index = threadIdx.x + blockIdx.x * blockDim.x;

    if (vertex_index >= row_pointers.size(0)) {
        return;
    }

    const auto [start, end] = edge_range(col_indices.size(0), row_pointers, vertex_index);

    float sum = 0.0;
    for (auto edge_index = start; edge_index < end; edge_index++) {
        sum += features[col_indices[edge_index]];
    }

    const auto degree = end - start;
    averages[vertex_index] = degree == 0 ? 0.0 : sum / degree;
}

void average_neighbor_features(
    torch::Tensor col_indices,
    torch::Tensor row_pointers,
    torch::Tensor features,
    torch::Tensor averages
) {
    constexpr auto block_size = 64;

    const auto thread_count = row_pointers.size(0);
    const auto block_count = div_ceil(thread_count, block_size);

    if (col_indices.scalar_type() == torch::ScalarType::Short) {
        vertex_average_neighbor_features<short><<<block_count, block_size>>>(
            col_indices.packed_accessor32<short, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            features.packed_accessor32<float,  1>(),
            averages.packed_accessor32<float,  1>()
        );
    } else if (col_indices.scalar_type() == torch::ScalarType::Int) {
        vertex_average_neighbor_features<int><<<block_count, block_size>>>(
            col_indices.packed_accessor32<int, 1>(),
            row_pointers.packed_accessor32<int, 1>(),
            features.packed_accessor32<float,  1>(),
            averages.packed_accessor32<float,  1>()
        );
    }

    hipDeviceSynchronize();

}
